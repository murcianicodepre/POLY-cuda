#include "hip/hip_runtime.h"
#include "Material.h"
#include "PolyRenderer.h"

/*
    Material ~ Material class
    Diego Párraga Nicolás ~ diegojose.parragan@um.es
*/

hipArray_t createTexture(const char* path, hipTextureObject_t& texObj){
    // Load texture into host memory
    uchar4* texData = reinterpret_cast<uchar4*>(PolyRenderer::loadPNG(path));

    // Channel descriptor and cuda Array
    hipArray_t texArray;
    hipChannelFormatDesc chanDesc = hipCreateChannelDesc<uchar4>();
    hipMallocArray(&texArray, &chanDesc, TEXTURE_SIZE, TEXTURE_SIZE);
    hipMemcpy2DToArray(texArray, 0,0, texData, TEXTURE_SIZE*sizeof(uchar4), TEXTURE_SIZE*sizeof(uchar4), TEXTURE_SIZE, hipMemcpyHostToDevice);
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = texArray;
    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = true;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    
    free(texData);

    return texArray;
}

Material::Material(float diff, float spec, float reflective, float refractive) : texture(0), bump(0), diff(diff), spec(spec), reflective(reflective), refractive(refractive) {}
hipArray_t Material::loadTexture(const char* path){ return createTexture(path, texture); }
hipArray_t Material::loadBump(const char* path){ return createTexture(path, bump); }